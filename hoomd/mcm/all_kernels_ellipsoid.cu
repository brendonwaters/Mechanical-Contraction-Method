// Copyright (c) 2009-2018 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "ComputeFreeVolumeGPU.cuh"
#include "IntegratorMCMMonoGPU.cuh"
#include "IntegratorMCMMonoImplicitGPU.cuh"
#include "IntegratorMCMMonoImplicitNewGPU.cuh"

#include "ShapeEllipsoid.h"

namespace mcm
{

namespace detail
{

//! MCM kernels for ShapeEllipsoid
template hipError_t gpu_mcm_free_volume<ShapeEllipsoid>(const mcm_free_volume_args_t &args,
                                                       const typename ShapeEllipsoid::param_type *d_params);
template hipError_t gpu_mcm_update<ShapeEllipsoid>(const mcm_args_t& args,
                                                  const typename ShapeEllipsoid::param_type *d_params);
template hipError_t gpu_mcm_implicit_count_overlaps<ShapeEllipsoid>(const mcm_implicit_args_t& args,
                                                  const typename ShapeEllipsoid::param_type *d_params);
template hipError_t gpu_mcm_implicit_accept_reject<ShapeEllipsoid>(const mcm_implicit_args_t& args,
                                                  const typename ShapeEllipsoid::param_type *d_params);
template hipError_t gpu_mcm_insert_depletants_queue<ShapeEllipsoid>(const mcm_implicit_args_new_t& args,
                                                  const typename ShapeEllipsoid::param_type *d_params);
template hipError_t gpu_mcm_implicit_accept_reject_new<ShapeEllipsoid>(const mcm_implicit_args_new_t& args,
                                                  const typename ShapeEllipsoid::param_type *d_params);

}; // end namespace detail

} // end namespace mcm
