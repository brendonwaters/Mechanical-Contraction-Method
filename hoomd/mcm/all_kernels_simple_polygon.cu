// Copyright (c) 2009-2018 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "ComputeFreeVolumeGPU.cuh"
#include "IntegratorHPMCMonoGPU.cuh"
#include "IntegratorHPMCMonoImplicitGPU.cuh"
#include "IntegratorHPMCMonoImplicitNewGPU.cuh"

#include "ShapeSimplePolygon.h"

namespace mcm
{

namespace detail
{

//! HPMC kernels for ShapeSimplePolygon
template hipError_t gpu_mcm_free_volume<ShapeSimplePolygon>(const mcm_free_volume_args_t &args,
                                                       const typename ShapeSimplePolygon::param_type *d_params);
template hipError_t gpu_mcm_update<ShapeSimplePolygon>(const mcm_args_t& args,
                                                  const typename ShapeSimplePolygon::param_type *d_params);
template hipError_t gpu_mcm_implicit_count_overlaps<ShapeSimplePolygon>(const mcm_implicit_args_t& args,
                                                  const typename ShapeSimplePolygon::param_type *d_params);
template hipError_t gpu_mcm_implicit_accept_reject<ShapeSimplePolygon>(const mcm_implicit_args_t& args,
                                                  const typename ShapeSimplePolygon::param_type *d_params);
template hipError_t gpu_mcm_insert_depletants_queue<ShapeSimplePolygon>(const mcm_implicit_args_new_t& args,
                                                  const typename ShapeSimplePolygon::param_type *d_params);
template hipError_t gpu_mcm_implicit_accept_reject_new<ShapeSimplePolygon>(const mcm_implicit_args_new_t& args,
                                                  const typename ShapeSimplePolygon::param_type *d_params);

}; // end namespace detail

} // end namespace mcm
