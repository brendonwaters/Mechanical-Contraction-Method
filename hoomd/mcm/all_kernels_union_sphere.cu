// Copyright (c) 2009-2018 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "ComputeFreeVolumeGPU.cuh"
#include "IntegratorMCMMonoGPU.cuh"
#include "IntegratorMCMMonoImplicitGPU.cuh"
#include "IntegratorMCMMonoImplicitNewGPU.cuh"

#include "ShapeUnion.h"

namespace mcm
{

namespace detail
{

//! MCM kernels for ShapeUnion<ShapeSphere>
template hipError_t gpu_mcm_free_volume<ShapeUnion<ShapeSphere> >(const mcm_free_volume_args_t &args,
                                                       const typename ShapeUnion<ShapeSphere> ::param_type *d_params);
template hipError_t gpu_mcm_update<ShapeUnion<ShapeSphere> >(const mcm_args_t& args,
                                                  const typename ShapeUnion<ShapeSphere> ::param_type *d_params);
template hipError_t gpu_mcm_implicit_count_overlaps<ShapeUnion<ShapeSphere> >(const mcm_implicit_args_t& args,
                                                  const typename ShapeUnion<ShapeSphere> ::param_type *d_params);
template hipError_t gpu_mcm_implicit_accept_reject<ShapeUnion<ShapeSphere> >(const mcm_implicit_args_t& args,
                                                  const typename ShapeUnion<ShapeSphere> ::param_type *d_params);
template hipError_t gpu_mcm_insert_depletants_queue<ShapeUnion<ShapeSphere> >(const mcm_implicit_args_new_t& args,
                                                  const typename ShapeUnion<ShapeSphere> ::param_type *d_params);
template hipError_t gpu_mcm_implicit_accept_reject_new<ShapeUnion<ShapeSphere> >(const mcm_implicit_args_new_t& args,
                                                  const typename ShapeUnion<ShapeSphere> ::param_type *d_params);

}; // end namespace detail

} // end namespace mcm
